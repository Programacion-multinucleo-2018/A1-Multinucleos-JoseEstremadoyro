
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void multiplyCell(int N,int * a, int * b, int * c){

        

}

void GPUTimedMatrixMultiplication(int N,int * a,int * b, int * c,

        int ** runs, int runsLength){

    // Allocate in GPU
    int *d_a,*d_b,*d_c;
    int size = N*sizeof(int);
    hipMalloc(&d_a,size);
    hipMalloc(&d_b,size);
    hipMalloc(&d_c,size);

    // Transfer to device
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

    // Declare time tables with corresponding data :todo

    // Call kernel with the blocks, grid and threads specified
    for(int i=0;i<runsLength;i++){

        int * run = runs[i];
        dim3 blocksPerGrid(run[0],run[1],run[2]);
        dim3 threadsPerBlock(run[3],run[4],run[5]);

        // Gather initial time :todo
        multiplyCell<<<blocksPerGrid,threadsPerBlock>>>(N,a,b,c); 
        // Gather finishing time :todo

    }

    // Print time table :todo

    // Free variables
    hipFree(d_a);
    hipFree(d_b);


}

void GPUMatrixMultiplication(int N,int * a,int * b, int * c,

        int ** runs, int runsLength){

    // Allocate in GPU
    int *d_a,*d_b,*d_c;
    int size = N*sizeof(int);
    hipMalloc(&d_a,size);
    hipMalloc(&d_b,size);
    hipMalloc(&d_c,size);

    // Transfer to device
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

    // Call kernel with the blocks, grid and threads specified
    for(int i=0;i<runsLength;i++){

        int * run = runs[i];
        dim3 blocksPerGrid(run[0],run[1],run[2]);
        dim3 threadsPerBlock(run[3],run[4],run[5]);
        multiplyCell<<<blocksPerGrid,threadsPerBlock>>>(N,a,b,c); 

    }

    // Free variables
    hipFree(d_a);
    hipFree(d_b);

}
