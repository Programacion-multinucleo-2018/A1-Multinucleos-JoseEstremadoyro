
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__
void multiplyCell(int N,int * a, int * b, int * c){

    // We get the index of the current data 
    unsigned int threadx = blockDim.x * blockIdx.x + threadIdx.x;

    // Then we get the col and row
    int row = threadx % N;
    int col = threadx / N;

    // Then we multiply and add each one of them
    int result = 0;
    /*for(int i=0;i<N;i++){
        //result +=a[row*N+i]+b[i*N+col];
    }*/
    result = a[threadx]+b[threadx];
    c[threadx]=result;

}

void GPUTimedMatrixMultiplication(int N,int * a,int * b, int * c,

        int ** runs, int runsLength){

    // Allocate in GPU
    int *d_a,*d_b,*d_c;
    int size = N*sizeof(int);
    hipMalloc(&d_a,size);
    hipMalloc(&d_b,size);
    hipMalloc(&d_c,size);

    // Transfer to device
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

    // Declare time tables with corresponding data :todo

    // Call kernel with the blocks, grid and threads specified
    for(int i=0;i<runsLength;i++){

        int * run = runs[i];
        dim3 blocksPerGrid(run[0],run[1],run[2]);
        dim3 threadsPerBlock(run[3],run[4],run[5]);

        // Gather initial time :todo
        multiplyCell<<<blocksPerGrid,threadsPerBlock>>>(N,a,b,c); 
        // Gather finishing time :todo

    }

    // Copy result back from gpu
    hipMemcpy(d_c,c,size,hipMemcpyDeviceToHost);

    // Print time table :todo

    // Free variables
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}

void GPUMatrixMultiplication(int N,int * a,int * b, int * c,
        int * run){

    // Allocate in GPU
    int *d_a,*d_b,*d_c;
    int size = N*sizeof(int);
    hipMalloc(&d_a,size);
    hipMalloc(&d_b,size);
    hipMalloc(&d_c,size);

    // Transfer to device
    hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);

    // Call kernel with the blocks, grid and threads specified
    dim3 blocksPerGrid(run[0],run[1],run[2]);
    dim3 threadsPerBlock(run[3],run[4],run[5]);
    multiplyCell<<<blocksPerGrid,threadsPerBlock>>>(N,d_a,d_b,d_c); 

    // Copy result back from gpu
    hipMemcpy(c,d_c,size,hipMemcpyDeviceToHost);

    // Free variables
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

}
