
#include <hip/hip_runtime.h>
#include <stdio.h>
void generateRuns(int ** runs,int * length){
    // GetGPUProperties
    hipDeviceProp_t props; 
    hipGetDeviceProperties(&props,0);

    // Get maximum threads, blocks and grids

    printf("GPU Info\n");
    printf("Name: %s\n",props.name);
    printf("Max Threads Per Block  %d\n",props.maxThreadsPerBlock);
    printf("Max Threads Size  %d %d %d\n",
            props.maxThreadsDim[0],
            props.maxThreadsDim[1],
            props.maxThreadsDim[2]);
    printf("Max Grid Size %d %d %d\n",
            props.maxGridSize[0],
            props.maxGridSize[1],
            props.maxGridSize[2]);
    printf("Compute Capability %d\n",props.major);
    // Get number of tests
    // Allocate runs
    // Generate the block, grid, threads
}
