
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
void generateRuns(int *** inputRuns,int * length,int N){

    // GetGPUProperties
    hipDeviceProp_t props; 
    hipGetDeviceProperties(&props,0);

    // Get maximum threads, blocks and grids
    printf("Generating test runs using N= %d:\n",N);
    printf("GPU Info\n");
    printf("Name: %s\n",props.name);
    printf("Max Threads Per Block  %d\n",props.maxThreadsPerBlock);
    printf("Max Threads Size  %d %d %d\n",
        props.maxThreadsDim[0],
        props.maxThreadsDim[1],
        props.maxThreadsDim[2]);
    printf("Max Grid Size %d %d %d\n",
        props.maxGridSize[0],
        props.maxGridSize[1],
        props.maxGridSize[2]);
    printf("Compute Capability %d\n",props.major);

    /* Get total number of tests- from linear to squared, 
       from threads to blocks */

    int t;
    if(N*N<props.maxThreadsPerBlock) 
        t = N*N;
    else
        t= props.maxThreadsPerBlock;
    int b = N*N/t+1;

    int threadsToBlocksTests = log2(t/32.0) + 1; 
    int linearToSquareTests = log2(t/1.0) + 1; 
    printf("%d %d\n",threadsToBlocksTests,linearToSquareTests);
    *length = threadsToBlocksTests + linearToSquareTests;

    // Allocate runs
    int ** runs = (int**)malloc((*length)*sizeof(int*));
    for(int i=0;i<*length;i++){
        runs[i] = (int*)malloc(6*sizeof(int));
    }
    
    // Generate the block, grid, threads
    // From linear to squared 
    int j = 0;
    int i;
    for(i=1;j<threadsToBlocksTests;i*=2){
       runs[j][0]= t/i; 
       runs[j][1]= 1; 
       runs[j][2]= 1; 
       runs[j][3]= b*i; 
       runs[j][4]= 1; 
       runs[j][5]= 1; 
       j++;
    }

    // From threads to blocks
    for(i=1;j<*length;i*=2){
       runs[j][0]= t/i; 
       runs[j][1]= i; 
       runs[j][2]= 1; 
       runs[j][3]= b; 
       runs[j][4]= 1; 
       runs[j][5]= 1; 
       j++;
    }
    *inputRuns = runs;

}

